/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

  http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#pragma once
#include <hip/hip_runtime.h>


#ifdef PADDLE_WITH_HETERPS
#include "paddle/fluid/framework/fleet/heter_ps/heter_comm_kernel.h"

namespace paddle {
namespace framework {

#ifdef PADDLE_WITH_CUDA

struct GPUCustomGradMerger {
  template <typename T>
  CUB_RUNTIME_FUNCTION __forceinline__ __device__ T
  operator()(const T& a, const T& b) const {
    T out;
    out.slot = a.slot;
    out.show = a.show + b.show;
    out.clk = a.clk + b.clk;
    out.lr_g = a.lr_g + b.lr_g;
    for (int i = 0; i < MF_DIM; ++i) {
      out.mf_g[i] = a.mf_g[i] + b.mf_g[i];
    }
    return out;
  }
} gpu_merger;

template <typename T>
__global__ void fill_idx_kernel(T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    idx[i] = i;
  }
}

// template <typename T>
// void show_tensor(T* input, size_t len, gpuStream_t stream, std::string
// name)
// {
//  T tmp[len];  // NOLINT
//  cudaMemcpyAsync(&tmp, input, sizeof(T) * len, cudaMemcpyDeviceToHost,
//  stream);
//  cudaStreamSynchronize(stream);
//  std::cout << name;
//  for (int i = 0; i < len; ++i) {
//    std::cout << ":" << tmp[i];
//  }
//  std::cout << std::endl;
//}

template <typename T>
__global__ void calc_shard_offset_kernel(T* idx, T* left, T* right,
                                         size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len - 1) {
    if (idx[i] != idx[i + 1]) {
      right[idx[i]] = i;
      left[idx[i + 1]] = i + 1;
    }
  }
  if (i == 0) {
    left[idx[i]] = i;
  }
  if (i == (len - 1)) {
    right[idx[i]] = i;
  }
}

template <typename KeyType, typename T>
__global__ void calc_shard_index_kernel(KeyType* d_keys, size_t len,
                                        T* shard_index, int total_gpu) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    shard_index[i] = d_keys[i] % total_gpu;
  }
}

template <typename KeyType, typename T>
__global__ void fill_shard_key_kernel(KeyType* d_shard_keys, KeyType* d_keys,
                                      T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_shard_keys[i] = d_keys[idx[i]];
  }
}

template <typename KeyType, typename GradType, typename T>
__global__ void fill_shard_grads_kernel(KeyType* d_shard_keys, KeyType* d_keys,
                                        GradType* d_shard_grads,
                                        GradType* d_grads, T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_shard_keys[i] = d_keys[idx[i]];
    d_shard_grads[i] = d_grads[idx[i]];
  }
}

template <typename ValType, typename T>
__global__ void fill_dvals_kernel(ValType* d_shard_vals, ValType* d_vals,
                                  T* idx, size_t len) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_vals[idx[i]] = d_shard_vals[i];
  }
}

template <typename KeyType, typename T>
__global__ void dy_mf_fill_shard_grads_kernel(
    KeyType* d_shard_keys, KeyType* d_keys, float* d_shard_grads,
    float* d_grads, T* idx, size_t len, size_t grad_value_size,
    CommonFeatureValueAccessor feature_value_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    d_shard_keys[i] = d_keys[idx[i]];
    float* cur = (float*)((char*)d_shard_grads + i * grad_value_size);
    float* shard_val = (float*)((char*)d_grads + uint64_t(idx[i]) * grad_value_size);

    cur[feature_value_accessor.common_push_value.SlotIndex()] =
      shard_val[feature_value_accessor.common_push_value.SlotIndex()];
    cur[feature_value_accessor.common_push_value.ShowIndex()] =
      shard_val[feature_value_accessor.common_push_value.ShowIndex()];
    cur[feature_value_accessor.common_push_value.ClickIndex()] =
      shard_val[feature_value_accessor.common_push_value.ClickIndex()];
    cur[feature_value_accessor.common_push_value.MfDimIndex()] =
      shard_val[feature_value_accessor.common_push_value.MfDimIndex()];
    cur[feature_value_accessor.common_push_value.EmbedGIndex()] =
      shard_val[feature_value_accessor.common_push_value.EmbedGIndex()];

    for (int x = 0; x < int(shard_val[feature_value_accessor.common_push_value.MfDimIndex()]); x++) {
      cur[feature_value_accessor.common_push_value.EmbedxGIndex() + x] = 
        shard_val[feature_value_accessor.common_push_value.EmbedxGIndex() + x];
    }
  }
}

template <typename KeyType>
__global__ void merge_gradients_basic_kernel(const KeyType* d_keys,
                                       const uint32_t* offset,
                                       const uint32_t* fea_num,
                                       const uint32_t* index, const char* input,
                                       char* output, int n,
                                       size_t grad_value_size,
                                       DynamicGradMerger& merger,
                                      CommonFeatureValueAccessor& feature_value_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    uint32_t start = offset[i];
    uint32_t num = fea_num[i];
    int ori_index = index[start];
    float* out = (float*)(output + i * grad_value_size);
    float* in =
        (float*)(input + size_t(ori_index) * grad_value_size);
    merger.update_basic(out, in, feature_value_accessor);
    KeyType key = d_keys[i];
    if (key != 0) {
      for (int j = 1; j < num; ++j) {
        ori_index = index[start + j];
        in = (float*)(input + size_t(ori_index) * grad_value_size);
        merger.merge_basic(out, in, feature_value_accessor);
      }
    }
  }
}

template <typename KeyType>
__global__ void merge_gradients_embedx_kernel(const KeyType* d_keys,
                                       const uint32_t* offset,
                                       const uint32_t* fea_num,
                                       const uint32_t* index, const char* input,
                                       char* output, int n,
                                       size_t grad_dim,
                                       size_t grad_value_size,
                                       DynamicGradMerger& merger,
                                      CommonFeatureValueAccessor& feature_value_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    size_t value_idx = i / grad_dim;
    size_t field_idx = i % grad_dim;
    uint32_t start = offset[value_idx];
    uint32_t num = fea_num[value_idx];
    int ori_index = index[start];
    float* in = (float*)(input + size_t(ori_index) * grad_value_size);
    float* out = (float*)(output + value_idx * grad_value_size);
    merger.update_embedx(out, in, field_idx, feature_value_accessor);
    KeyType key = d_keys[value_idx];
    if (key != 0) {
      for (int j = 1; j < num; ++j) {
        int ori_index = index[start + j];
        float* in = (float*)(input + size_t(ori_index) * grad_value_size);
        merger.merge_embedx(out, in, field_idx, feature_value_accessor);
      }
    }
  }
}

__global__ void split_segments_kernel(
        const uint32_t* d_fea_num_info, size_t n,
        uint32_t* d_segments, uint32_t* d_segments_num,
        uint32_t segment_size) {
  const size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tx >= n) {
    return;
  }

  auto fea_num = d_fea_num_info[tx];
  auto seg_num = (uint32_t)((fea_num - 1) / segment_size + 1);
  d_segments[tx] = seg_num;
}

__global__ void expand_segments_kernel(
        const uint32_t* d_fea_num_info,
        const uint32_t* d_segments_offset, size_t n,
        uint32_t* d_segments_fea_num_info, uint32_t segment_size) {
  const size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tx >= n) {
    return;
  }

  auto fea_num = d_fea_num_info[tx];
  auto seg_num = (uint32_t)((fea_num - 1) / segment_size + 1);
  auto start_pos = d_segments_offset[tx];
  auto remains = fea_num;
  int cur_seg_size = 0;
  for (size_t i = 0; i < seg_num; ++i) {
    if (remains >= segment_size) {
      cur_seg_size = segment_size;
    } else {
      cur_seg_size = remains;
    }
    d_segments_fea_num_info[start_pos + i] = cur_seg_size;
    remains -= cur_seg_size;
  }
}

template <typename KeyType>
__global__ void shrink_keys_kernel(
        const KeyType* d_keys, const uint32_t* d_segments_offset,
        KeyType* d_segments_keys, size_t n) {
  const size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tx >= n) {
    return;
  }

  d_segments_keys[tx] = d_keys[d_segments_offset[tx]];
}

template<typename KeyType, typename T>
__global__ void fill_restore_idx_kernel(
        const T *d_sorted_idx,
        const T *d_offset,
        const T *d_merged_cnts,
        const KeyType *d_merged_keys,
        T *d_restore_idx,
        size_t n) {
  const size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tx >= n) {
    return;
  }

  const KeyType & key = d_merged_keys[tx];
  if (key == 0) {
    return;
  }

  const T &off = d_offset[tx];
  const T &num = d_merged_cnts[tx];
  for (size_t k = 0; k < num; ++k) {
    d_restore_idx[d_sorted_idx[off + k]] = tx;
  }
}

template<typename KeyType>
__global__ void unpack_merged_vals_kernel(
        const KeyType* d_keys,
        const float* d_merged_vals,
        const uint32_t* d_restored_idx,
        float* d_out, size_t val_size, const size_t n,
        CommonFeatureValueAccessor feature_value_accessor) {
  const size_t tx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tx >= n) {
    return;
  }

  size_t src_val_idx = 0;
  const KeyType & key = d_keys[tx];
  if (key != 0) {
    src_val_idx = d_restored_idx[tx];
  }

  uint64_t dst_offset = uint64_t(tx) * val_size;
  float* dst = (float*)((char*)d_out + dst_offset);
  float* src_val = (float*)((char*)d_merged_vals + uint64_t(src_val_idx) * val_size);
  int mf_dim = int(src_val[feature_value_accessor.common_feature_value.MfDimIndex()]);

  *(reinterpret_cast<uint64_t*>(dst + feature_value_accessor.common_feature_value.CpuPtrIndex())) =
      *(reinterpret_cast<uint64_t*>(src_val + feature_value_accessor.common_feature_value.CpuPtrIndex()));
  dst[feature_value_accessor.common_feature_value.DeltaScoreIndex()] =
      src_val[feature_value_accessor.common_feature_value.DeltaScoreIndex()];
  dst[feature_value_accessor.common_feature_value.ShowIndex()] =
      src_val[feature_value_accessor.common_feature_value.ShowIndex()];
  dst[feature_value_accessor.common_feature_value.ClickIndex()] =
      src_val[feature_value_accessor.common_feature_value.ClickIndex()];
  dst[feature_value_accessor.common_feature_value.EmbedWIndex()] =
      src_val[feature_value_accessor.common_feature_value.EmbedWIndex()];
  for (int i = 0; i < feature_value_accessor.common_feature_value.EmbedDim(); i++) {
      dst[feature_value_accessor.common_feature_value.EmbedG2SumIndex() + i] = 
          src_val[feature_value_accessor.common_feature_value.EmbedG2SumIndex() + i];
  }
  dst[feature_value_accessor.common_feature_value.SlotIndex()] =
      src_val[feature_value_accessor.common_feature_value.SlotIndex()];
  dst[feature_value_accessor.common_feature_value.MfDimIndex()] = mf_dim;
  dst[feature_value_accessor.common_feature_value.MfSizeIndex()] =
      src_val[feature_value_accessor.common_feature_value.MfSizeIndex()];

  for (int x = feature_value_accessor.common_feature_value.EmbedxG2SumIndex();
          x < int(feature_value_accessor.common_feature_value.Size(mf_dim) / sizeof(float)); x++){
    dst[x] = src_val[x];
  }
}

template <typename T>
__global__ void dy_mf_fill_dvals_kernel(float* d_shard_vals, float* d_vals,
                                        T* idx, size_t len, size_t val_size,
                                       CommonFeatureValueAccessor feature_value_accessor) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    uint64_t new_offset = uint64_t(idx[i]) * val_size;
    float* cur = (float*)((char*)d_vals + new_offset);
    float* shard_val = (float*)((char*)d_shard_vals + uint64_t(i) * val_size);
    int mf_dim = int(shard_val[feature_value_accessor.common_feature_value.MfDimIndex()]);

    *(reinterpret_cast<uint64_t*>(cur + feature_value_accessor.common_feature_value.CpuPtrIndex())) =
      *(reinterpret_cast<uint64_t*>(shard_val + feature_value_accessor.common_feature_value.CpuPtrIndex()));
    cur[feature_value_accessor.common_feature_value.DeltaScoreIndex()] =
      shard_val[feature_value_accessor.common_feature_value.DeltaScoreIndex()];
    cur[feature_value_accessor.common_feature_value.ShowIndex()] =
      shard_val[feature_value_accessor.common_feature_value.ShowIndex()];
    cur[feature_value_accessor.common_feature_value.ClickIndex()] =
      shard_val[feature_value_accessor.common_feature_value.ClickIndex()];
    cur[feature_value_accessor.common_feature_value.EmbedWIndex()] =
      shard_val[feature_value_accessor.common_feature_value.EmbedWIndex()];
    for (int i = 0; i < feature_value_accessor.common_feature_value.EmbedDim(); i++) {
      cur[feature_value_accessor.common_feature_value.EmbedG2SumIndex() + i] = 
        shard_val[feature_value_accessor.common_feature_value.EmbedG2SumIndex() + i];
    }
    cur[feature_value_accessor.common_feature_value.SlotIndex()] =
      shard_val[feature_value_accessor.common_feature_value.SlotIndex()];
    cur[feature_value_accessor.common_feature_value.MfDimIndex()] = mf_dim;
    cur[feature_value_accessor.common_feature_value.MfSizeIndex()] =
      shard_val[feature_value_accessor.common_feature_value.MfSizeIndex()];

    for (int x = feature_value_accessor.common_feature_value.EmbedxG2SumIndex();
            x < int(feature_value_accessor.common_feature_value.Size(mf_dim) / sizeof(float)); x++){
      cur[x] = shard_val[x];
    }
  }
}

// cuda implemention of  heter_comm_kernel.h
template <typename T, typename StreamType>
void HeterCommKernel::fill_idx(T* idx, long long len,
                               const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_idx_kernel<<<grid_size, block_size_, 0, stream>>>(idx, c_len);
}

template <typename T, typename StreamType>
void HeterCommKernel::calc_shard_offset(T* idx, T* left, T* right,
                                        long long len, int total_devs,
                                        const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  calc_shard_offset_kernel<<<grid_size, block_size_, 0, stream>>>(idx, left,
                                                                  right, c_len);
}

template <typename KeyType, typename T, typename StreamType>
void HeterCommKernel::calc_shard_index(KeyType* d_keys, long long len,
                                       T* shard_index, int total_gpu,
                                       const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  calc_shard_index_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_keys, c_len, shard_index, total_gpu);
}

template <typename KeyType, typename T, typename StreamType>
void HeterCommKernel::fill_shard_key(KeyType* d_shard_keys, KeyType* d_keys,
                                     T* idx, long long len,
                                     const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_shard_key_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_shard_keys, d_keys, idx, c_len);
}

template <typename KeyType, typename GradType, typename T, typename StreamType>
void HeterCommKernel::fill_shard_grads(KeyType* d_shard_keys, KeyType* d_keys,
                                       GradType* d_shard_grads,
                                       GradType* d_grads, T* idx, long long len,
                                       const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_shard_grads_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_shard_keys, d_keys, d_shard_grads, d_grads, idx, c_len);
}

template <typename ValType, typename T, typename StreamType>
void HeterCommKernel::fill_dvals(ValType* d_shard_vals, ValType* d_vals, T* idx,
                                 long long len, const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  fill_dvals_kernel<<<grid_size, block_size_, 0, stream>>>(d_shard_vals, d_vals,
                                                           idx, c_len);
}

template <typename KeyT, typename ValueT, typename StreamType>
void HeterCommKernel::sort_pairs(void* d_temp_storage,
                                 size_t& temp_storage_bytes,  // NOLINT
                                 const KeyT* d_keys_in,       // NOLINT
                                 KeyT* d_keys_out, const ValueT* d_values_in,
                                 ValueT* d_values_out, int num_items,
                                 int begin_bit, int end_bit, StreamType stream,
                                 bool debug_synchronous) {
  PADDLE_ENFORCE_GPU_SUCCESS(cub::DeviceRadixSort::SortPairs(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in,
      d_values_out, num_items, begin_bit, end_bit, stream, debug_synchronous));
}

template <typename KeysInputIteratorT, typename UniqueOutputIteratorT,
          typename ValuesInputIteratorT, typename AggregatesOutputIteratorT,
          typename NumRunsOutputIteratorT, typename StreamType>
void HeterCommKernel::reduce_by_key(void* d_temp_storage,
                                    size_t& temp_storage_bytes,  // NOLINT
                                    KeysInputIteratorT d_keys_in,
                                    UniqueOutputIteratorT d_unique_out,
                                    ValuesInputIteratorT d_values_in,
                                    AggregatesOutputIteratorT d_aggregates_out,
                                    NumRunsOutputIteratorT d_num_runs_out,
                                    int num_items, StreamType stream,
                                    bool debug_synchronous) {
  PADDLE_ENFORCE_GPU_SUCCESS(cub::DeviceReduce::ReduceByKey(
      d_temp_storage, temp_storage_bytes, d_keys_in, d_unique_out, d_values_in,
      d_aggregates_out, d_num_runs_out, gpu_merger, num_items, stream,
      debug_synchronous));
}

template <typename KeyType, typename T, typename StreamType>
void HeterCommKernel::dy_mf_fill_shard_grads(
    KeyType* d_shard_keys, KeyType* d_keys, float* d_shard_grads,
    float* d_grads, T* idx, long long len, size_t grad_value_size,
    const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  dy_mf_fill_shard_grads_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_shard_keys, d_keys, d_shard_grads, d_grads, idx, c_len,
      grad_value_size, feature_value_accessor_);
}

template <typename KeyType, typename StreamType>
void HeterCommKernel::merge_gradient(
    const KeyType* d_keys,
    const uint32_t* offset, const uint32_t* fea_num, const uint32_t* index,
    const char* input, char* output, int n, size_t grad_dim, size_t grad_value_size,
    DynamicGradMerger& merger, const StreamType& stream) {
  int grid_size1 = (n - 1) / block_size_ + 1;
  merge_gradients_basic_kernel<<<grid_size1, block_size_, 0, stream>>>(
      d_keys,
      offset, fea_num, index, input, output, n, grad_value_size, merger, feature_value_accessor_);
  if (grad_dim > 0) {
    int grid_size2 = (n * grad_dim - 1) / block_size_ + 1;
    merge_gradients_embedx_kernel<<<grid_size2, block_size_, 0, stream>>>(
            d_keys,
            offset, fea_num, index, input, output, n * grad_dim, grad_dim, grad_value_size, merger, feature_value_accessor_);
  }
}

template <typename T, typename StreamType>
void HeterCommKernel::dy_mf_fill_dvals(float* d_shard_vals, float* d_vals,
                                       T* idx, long long len, size_t val_size,
                                       const StreamType& stream) {
  int grid_size = (len - 1) / block_size_ + 1;
  size_t c_len = (size_t)len;
  dy_mf_fill_dvals_kernel<<<grid_size, block_size_, 0, stream>>>(
      d_shard_vals, d_vals, idx, c_len, val_size, feature_value_accessor_);
}

template <typename StreamType>
void HeterCommKernel::split_segments(const uint32_t* d_fea_num_info, size_t n,
        uint32_t* d_segments, uint32_t* d_segments_num, size_t segment_size, const StreamType& stream) {
  int grid_size = (n - 1) / block_size_ + 1;
  split_segments_kernel<<<grid_size, block_size_, 0, stream>>>(
          d_fea_num_info, n, d_segments, d_segments_num, segment_size);
}

template <typename StreamType>
void HeterCommKernel::expand_segments(const uint32_t* d_fea_num_info,
          const uint32_t* d_segments_offset, size_t n,
          uint32_t* d_segments_fea_num_info, uint32_t segment_size,
          const StreamType& stream) {
  int grid_size = (n - 1) / block_size_ + 1;
  expand_segments_kernel<<<grid_size, block_size_, 0, stream>>>(
          d_fea_num_info,
          d_segments_offset, n,
          d_segments_fea_num_info, segment_size);
}

template <typename KeyType, typename StreamType>
void HeterCommKernel::shrink_keys(const KeyType* d_keys, const uint32_t* d_segments_offset,
          KeyType* d_segments_keys, size_t n, const StreamType& stream) {
  int grid_size = (n - 1) / block_size_ + 1;
  shrink_keys_kernel<<<grid_size, block_size_, 0, stream>>>(
          d_keys, d_segments_offset, d_segments_keys, n);
}

template <typename KeyType, typename StreamType>
void HeterCommKernel::fill_restore_idx(
        const uint32_t* d_sorted_idx, const uint32_t* d_offset,
        const uint32_t* d_merged_cnts, const KeyType* d_merged_keys,
        const size_t n, uint32_t *d_restore_idx, const StreamType& stream) {
  int grid_size = (n - 1) / block_size_ + 1;
  fill_restore_idx_kernel<<<grid_size, block_size_, 0, stream>>>(
          d_sorted_idx, d_offset, d_merged_cnts, d_merged_keys, d_restore_idx, n);
}

template <typename KeyType, typename StreamType>
void HeterCommKernel::unpack_merged_vals(size_t n, const KeyType* d_keys,
        const void* d_merged_vals, const uint32_t* d_restore_idx,
        void* d_vals, size_t val_size, const StreamType& stream) {
  int grid_size = (n - 1) / block_size_ + 1;
  unpack_merged_vals_kernel<<<grid_size, block_size_, 0, stream>>>(
          d_keys, (const float *)d_merged_vals, d_restore_idx,
          (float *)d_vals, val_size, n, feature_value_accessor_);
}

template void HeterCommKernel::fill_idx<int, cudaStream_t>(
    int* idx, long long len, const cudaStream_t& stream);
template void HeterCommKernel::fill_idx<uint32_t, cudaStream_t>(
    uint32_t* idx, long long len, const cudaStream_t& stream);

template void HeterCommKernel::calc_shard_offset<int, cudaStream_t>(
    int* idx, int* left, int* right, long long len, int total_devs,
    const cudaStream_t& stream);
template void HeterCommKernel::calc_shard_index<
    unsigned long, int, cudaStream_t>(unsigned long* d_keys, long long len,
                                      int* shard_index, int total_devs,
                                      const cudaStream_t& stream);

template void HeterCommKernel::calc_shard_index<long, int, cudaStream_t>(
    long* d_keys, long long len, int* shard_index, int total_devs,
    const cudaStream_t& stream);

template void HeterCommKernel::fill_shard_key<long, int, cudaStream_t>(
    long* d_shard_keys, long* d_keys, int* idx, long long len,
    const cudaStream_t& stream);

template void HeterCommKernel::fill_shard_key<unsigned long, int, cudaStream_t>(
    unsigned long* d_shard_keys, unsigned long* d_keys, int* idx, long long len,
    const cudaStream_t& stream);

template void HeterCommKernel::fill_shard_grads<
    unsigned long, float, int, cudaStream_t>(
    unsigned long* d_shard_keys, unsigned long* d_keys,
    float* d_shard_grads,
    float* d_grads, int* idx, long long len,
    const cudaStream_t& stream);

template void
HeterCommKernel::fill_dvals<paddle::framework::FeatureValue, int, cudaStream_t>(
    paddle::framework::FeatureValue* d_shard_vals,
    paddle::framework::FeatureValue* d_vals, int* idx, long long len,
    const cudaStream_t& stream);

template void HeterCommKernel::sort_pairs<
    unsigned long, paddle::framework::FeaturePushValue, cudaStream_t>(
    void* d_temp_storage,
    size_t& temp_storage_bytes,      // NOLINT
    const unsigned long* d_keys_in,  // NOLINT
    unsigned long* d_keys_out,
    const paddle::framework::FeaturePushValue* d_values_in,
    paddle::framework::FeaturePushValue* d_values_out, int num_items,
    int begin_bit, int end_bit, cudaStream_t stream, bool debug_synchronous);

template void HeterCommKernel::sort_pairs<int, int, cudaStream_t>(
    void* d_temp_storage,
    size_t& temp_storage_bytes,  // NOLINT
    const int* d_keys_in,        // NOLINT
    int* d_keys_out, const int* d_values_in, int* d_values_out, int num_items,
    int begin_bit, int end_bit, cudaStream_t stream, bool debug_synchronous);

template void HeterCommKernel::reduce_by_key<
    unsigned long*, unsigned long*, paddle::framework::FeaturePushValue*,
    paddle::framework::FeaturePushValue*, int*, cudaStream_t>(
    void* d_temp_storage,
    size_t& temp_storage_bytes,  // NOLINT
    unsigned long* d_keys_in, unsigned long* d_unique_out,
    paddle::framework::FeaturePushValue* d_values_in,
    paddle::framework::FeaturePushValue* d_aggregates_out, int* d_num_runs_out,
    int num_items, cudaStream_t stream, bool debug_synchronous);

template void HeterCommKernel::dy_mf_fill_shard_grads<
    unsigned long, int, cudaStream_t>(
    unsigned long* d_shard_keys, unsigned long* d_keys,
    float* d_shard_grads, float* d_grads, int* idx, long long len,
    size_t grad_value_size, const cudaStream_t& stream);

template void HeterCommKernel::merge_gradient<uint32_t, cudaStream_t>(
    const uint32_t* d_keys,
    const uint32_t* offset, const uint32_t* fea_num, const uint32_t* index,
    const char* input, char* output, int n, size_t grad_dim, size_t grad_value_size,
    DynamicGradMerger& merger_, const cudaStream_t& stream);

template void HeterCommKernel::merge_gradient<uint64_t, cudaStream_t>(
    const uint64_t* d_keys,
    const uint32_t* offset, const uint32_t* fea_num, const uint32_t* index,
    const char* input, char* output, int n, size_t grad_dim, size_t grad_value_size,
    DynamicGradMerger& merger_, const cudaStream_t& stream);

template void HeterCommKernel::dy_mf_fill_dvals<int, cudaStream_t>(
    float* d_shard_vals,
    float* d_vals, int* idx, long long len,
    size_t val_size, const cudaStream_t& stream);

template void HeterCommKernel::split_segments<cudaStream_t>(
    const uint32_t* d_fea_num_info, size_t n,
    uint32_t* d_segment, uint32_t* d_segments_num, size_t segment_size,
    const cudaStream_t& stream);

template void HeterCommKernel::expand_segments<cudaStream_t>(
    const uint32_t* d_fea_num_info,
    const uint32_t* d_segments_offset, size_t n,
    uint32_t* d_segments_fea_num_info, uint32_t segment_size,
    const cudaStream_t& stream);

template void HeterCommKernel::shrink_keys<uint32_t, cudaStream_t>(
        const uint32_t* d_keys, const uint32_t* d_segments_offset,
        uint32_t* d_segments_keys, size_t segment_num, const cudaStream_t& stream);

template void HeterCommKernel::shrink_keys<uint64_t, cudaStream_t>(
        const uint64_t* d_keys, const uint32_t* d_segments,
        uint64_t* d_segments_keys, size_t total_segment_num, const cudaStream_t& stream);

template void HeterCommKernel::fill_restore_idx<uint64_t, cudaStream_t>(
        const uint32_t* d_sorted_idx, const uint32_t* d_offset,
        const uint32_t* d_merged_cnts, const uint64_t* d_merged_keys,
        const size_t n, uint32_t* d_restore_idx, const cudaStream_t& stream);

template void HeterCommKernel::fill_restore_idx<uint32_t, cudaStream_t>(
        const uint32_t* d_sorted_idx, const uint32_t* d_offset,
        const uint32_t* d_merged_cnts, const uint32_t* d_merged_keys,
        const size_t n, uint32_t* d_restore_idx, const cudaStream_t& stream);

template void HeterCommKernel::unpack_merged_vals<uint64_t, cudaStream_t>(
        size_t n, const uint64_t* d_keys, const void* d_merged_vals,
        const uint32_t* d_restore_idx, void* d_vals, size_t val_size,
        const cudaStream_t& stream);

template void HeterCommKernel::unpack_merged_vals<uint32_t, cudaStream_t>(
        size_t n, const uint32_t* d_keys, const void* d_merged_vals,
        const uint32_t* d_restore_idx, void* d_vals, size_t val_size,
        const cudaStream_t& stream);
#endif

}  // namespace framework
}  // namespace paddle
#endif
